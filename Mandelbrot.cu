#include "hip/hip_runtime.h"
// this program creates a .bmp file in the working directory

#include <math.h>
#include <fstream>
#include <windows.h> // contains windef.h which has all the bitmap stuff
#include <stdio.h> // defines FILENAME_MAX
#include <direct.h>

using namespace std;

// dimensions are hard coded
#define WIDTH 4096
#define HEIGHT 4096

// this kernel calculates the pixel value for one pixel
__global__ void mandelbrot(BYTE* imageData, float unitX, float unitY, int max, int pixelWidth)
{
	// get the unique thread index
	// only using 1, 1 grid
    int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	// offset values so center is 0, 0
	float offsetWidth = col - (WIDTH / 2);
	float offsetHeight = row - (HEIGHT / 2);

	// multiply by our units (applies the zoom)
	float translatedWidth = offsetWidth * unitX;
	float translatedHeight = offsetHeight * unitY;

	float x = 0, y = 0;
	int iter = 0;

	int pos = (WIDTH * row) + col; // the position in the pixel data byte array

	// keep iterating until point escapes mandlebrot set
	while (1)
	{
		if (sqrt((x*x) + (y*y)) > 2) // if magnitude is greater than 2
		{
			// point has escaped mandlebrot set - paint white
			imageData[pos * pixelWidth] = (BYTE)255;
			break;
		}
		if (iter == max)
		{
			// point is in the mandlebrot set - paint black
			imageData[pos * pixelWidth] = (BYTE)0;
			break;
		}

		// this applies the mandelbrot equation
		// Zn+1 = Zn^2 + C
		float x_new = ((x*x) - (y*y)) + translatedWidth;
		y = (2 * x*y) + translatedHeight;
		x = x_new;
		iter++;
	}
}

int main(int argc, char* argv[])
{
	printf("Building image data...\n");

    // this is hard coded sadly
	dim3 grid(256, 256);
	dim3 block(16, 16);

	int pixelWidth = 1; // in bytes. bmp doesn't really do binary images so 1 byte is minimum
	int imageSize = WIDTH * HEIGHT * pixelWidth; // in bytes

	// allocate device memory
	BYTE * imageData_d = NULL;
	hipMalloc((void **)&imageData_d, imageSize);

	// the interesting stuff in the mandlebrot set occurs between -2,-2 and 2,2
	float zoomX = 2, zoomY = 2;

	// max iterations
	// increasing iterations improves image quality but hits performance
	int max = 100;

	float unitX = zoomX / (WIDTH / 2);
	float unitY = zoomY / (HEIGHT / 2);

	// launch kernel on each pixel
	mandelbrot<<<grid, block>>>(imageData_d, unitX, unitY, max, pixelWidth);

	// copy data back to host
	BYTE * imageData_h = (BYTE*)malloc(imageSize);
	hipMemcpy(imageData_h, imageData_d, imageSize, hipMemcpyDeviceToHost);

    // construct the bitmap info header (DIB header)
	BITMAPINFOHEADER bmpInfoHeader = { 0 };
	bmpInfoHeader.biSize = sizeof(BITMAPINFOHEADER); // should be 40 bytes
	bmpInfoHeader.biHeight = HEIGHT;
	bmpInfoHeader.biWidth = WIDTH;
	bmpInfoHeader.biPlanes = 1; // number of color planes (always 1)
	bmpInfoHeader.biBitCount = pixelWidth * 8;
	bmpInfoHeader.biCompression = BI_RGB; // do not compress
	bmpInfoHeader.biSizeImage = imageSize; // image size in bytes
	bmpInfoHeader.biClrUsed = 0; // no colors
	bmpInfoHeader.biClrImportant = 0; // all colors important

	// construct bitmap file header
	BITMAPFILEHEADER bfh;
	bfh.bfType = 0x4D42; // the first two bytes of the file are 'BM' in ASCII, in little endian
	bfh.bfOffBits = sizeof(BITMAPINFOHEADER) + sizeof(BITMAPFILEHEADER) + (sizeof(RGBQUAD) * 256); // the offset (starting address of pixel data). size of headers + color table
	bfh.bfSize = bfh.bfOffBits + bmpInfoHeader.biSizeImage; // total size of image including size of headers

	// create the color table
	RGBQUAD colorTable[256];
	for (int i = 0; i < 256; i++)
	{
		colorTable[i].rgbBlue = (BYTE)i;
		colorTable[i].rgbGreen = (BYTE)i;
		colorTable[i].rgbRed = (BYTE)i;
		colorTable[i].rgbReserved = (BYTE)i;
	}

	// write everything to file
	ofstream imageFile;

	char filePath[FILENAME_MAX];
	// get the current working directory
	if (!_getcwd(filePath, FILENAME_MAX))
	{
		printf("error accessing current working directory\n");
		return 0;
	}

	printf("The current working directory is %s\n", filePath);
	strcat_s(filePath, "\\mandelbrot.bmp"); // append the image file name

	imageFile.open(filePath);
	imageFile.write((char *)&bfh, sizeof(bfh)); // Write the File header
	imageFile.write((char *)&bmpInfoHeader, sizeof(bmpInfoHeader)); // Write the bitmap info header
	imageFile.write((char *)&colorTable, sizeof(RGBQUAD) * 256); // Write the color table

	// if number of rows is a multiple of 4 bytes
	if (WIDTH % 4 == 0)
	{
		// write the image judata
		imageFile.write((char*)imageData_h, bmpInfoHeader.biSizeImage);
	}
	else
	{
		// else write and pad each row out with empty bytes
		char* padding = new char[4 - WIDTH % 4];
		for (int i = 0; i < HEIGHT; ++i)
		{
			imageFile.write((char *)&imageData_h[i * WIDTH], WIDTH);
			imageFile.write((char *)padding, 4 - WIDTH % 4);
		}
	}

	imageFile.close();
	printf("image file saved to %s\n", filePath);

	// clean up
	hipDeviceReset();
	hipFree(imageData_d);
	free(imageData_h);

	return 0;
}
